
#include <hip/hip_runtime.h>
//#include "raytrac_sdl2.h"

/*

// Minimum ve maksimum ���k uzakl�klar�
const double min_distance = 0.001;
const double max_distance = 1000.0;

__device__ Vec3SIMD calculate_light_contribution(const Light* light, const Vec3SIMD& point, const Vec3SIMD& normal) {
    Vec3SIMD to_light = light->position - point;
    double distance = to_light.length();

    if (distance < min_distance) distance = min_distance;
    if (distance > max_distance) return Vec3SIMD(0, 0, 0);

    to_light = to_light.normalize();
    double cos_theta = max(0.0, Vec3::dot(normal, to_light));

    Vec3SIMD intensity = light->intensity;

    if (auto point_light = dynamic_cast<const PointLight*>(light)) {
        // Inverse square law for point lights
        intensity = intensity / (distance * distance);
    }
    else if (auto directional_light = dynamic_cast<const DirectionalLight*>(light)) {
        // Constant intensity for directional lights
        // We do nothing here as intensity is already constant
    }
    else {
        // Default to inverse square law for other light types
        intensity = intensity / (distance * distance);
    }

    // Atmospheric effect (optional)
    double atmospheric_attenuation = exp(-distance * 0.0001);
    intensity = intensity * atmospheric_attenuation;

    return intensity * cos_theta;
}

__global__ void cuda_ray_color_kernel(const Ray* rays, const Hittable** world, const Light** lights, const Vec3SIMD* background_color, Vec3SIMD* final_colors, int max_depth, int num_rays, int num_lights) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_rays) return;

    Vec3SIMD final_color(0, 0, 0);
    Vec3SIMD throughput(1, 1, 1);
    Ray current_ray = rays[index];
    Ray scattered;
    Vec3 attenuation;

    for (int depth = 0; depth < max_depth; ++depth) {
        HitRecord rec;

        // Check for intersection with both objects and volumetric materials
        if (!world[0]->hit(current_ray, 0.001, infinity, rec)) {
            final_color += throughput * (*background_color);
            break;
        }

        if (rec.material->type() == MaterialType::Volumetric) {
            auto volumetric = std::dynamic_pointer_cast<Volumetric>(rec.material);
            if (!volumetric) {
                // Hata durumu veya i�leme devam edilemeyecek durum.
                break;
            }

            if (volumetric->scatter(current_ray, rec, attenuation, scattered)) {
                throughput *= Vec3SIMD(attenuation);
                current_ray = scattered;
                continue;
            }
            else {
                break;
            }
        }
        else {
            Vec3SIMD emitted = Vec3SIMD(rec.material->emitted(rec.u, rec.v, rec.point));
            final_color += throughput * emitted;

            if (!rec.material->scatter(current_ray, rec, attenuation, scattered)) {
                // Error condition
                break;
            }

            if (rec.material->type() != MaterialType::Dielectric && rec.material->type() != MaterialType::Volumetric) {
                Vec3SIMD direct_light(0, 0, 0);
                for (int i = 0; i < num_lights; ++i) {
                    Vec3SIMD light_contribution = calculate_light_contribution(lights[i], rec.point, rec.normal);
                    direct_light += light_contribution;
                }
                final_color += throughput * Vec3SIMD(attenuation) * direct_light;
            }

            throughput *= Vec3SIMD(attenuation);
            float p = std::max(0.1f, std::min(0.95f, throughput.max_component()));
            if (random_double() >= p) {
                break;
            }
            throughput /= p;
            current_ray = scattered;
        }
    }

    final_colors[index] = final_color;
}

void cuda_ray_color(const Ray* rays,
    Hittable** world,  // const kald�r�ld�
    Light** lights,    // const kald�r�ld�
    const Vec3SIMD& background_color,
    Vec3SIMD* final_colors,
    int max_depth,
    int num_rays,
    int num_lights) {
    // CUDA kernel call parameters
    int threads_per_block = 256;
    int blocks_per_grid = (num_rays + threads_per_block - 1) / threads_per_block;

    // CUDA kernel function launch
    cuda_ray_color_kernel << < blocks_per_grid, threads_per_block >> > (rays, world, lights, &background_color, final_colors, max_depth, num_rays, num_lights);

    // CUDA synchronization and error checking
    cudaDeviceSynchronize();
    cudaError_t err = cudaGetLastError();
    if (err != cudaSuccess) {
        fprintf(stderr, "CUDA error in cuda_ray_color_kernel: %s\n", cudaGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}*/
